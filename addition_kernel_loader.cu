#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int sum_host(const char* source, float* aDev, float* bDev, float* cDev, float *c, int n)
{
	int nb = n * sizeof(float);
	void* args[] = { (void*)&aDev, (void*)&bDev, (void*)&cDev };
	// Load module.
	hipModule_t module;
	hipError_t cuerr;
	cuerr = hipModuleLoad(&module, source);
	assert(cuerr == hipSuccess);

	// Load kernel.
	hipFunction_t kernel;
	cuerr = hipModuleGetFunction(&kernel, module, "addition_kernel");
	assert(cuerr == hipSuccess);
	
	// Launch kernel.
	cuerr = hipModuleLaunchKernel(kernel,
		n / BLOCK_SIZE, 1, 1, BLOCK_SIZE, 1, 1, 512,
		0, args, NULL);

	// Check error status from the launched kernel.
	hipError_t cudaerr = hipGetLastError();
	// Wait for kernel completion.
	cudaerr = hipDeviceSynchronize();

	// Copy the resulting array back to the host memory.
	cudaerr = hipMemcpy(c, cDev, nb, hipMemcpyDeviceToHost);

	return 1;
}

#include <malloc.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

double getTime()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);
	double seconds = tv.tv_sec*1000000;
	double milliseconds = tv.tv_usec;
	//printf("%lf\n", milliseconds);
	return (seconds+milliseconds);
}

int main ( int argc, char* argv[] )
{
	long long power = 1;
	int n = 512000000, nb = n * sizeof(float);
	float* a = (float*)malloc(nb);
	float* b = (float*)malloc(nb);
	float* c = (float*)malloc(nb);
	double idrandmax = 1.0 / RAND_MAX;

	float* aDev = NULL;
	float* bDev = NULL;
	float* cDev = NULL;
	
	int result = 0;	


	hipError_t cuerr;

	// Allocate memory on the GPU.
	hipError_t cudaerr = hipMalloc((void**)&aDev, nb);
	cudaerr = hipMalloc((void**)&bDev, nb);
	cudaerr = hipMalloc((void**)&cDev, nb);

	// Copy input data to device memory.
	cudaerr = hipMemcpy(aDev, a, nb, hipMemcpyHostToDevice);
	cudaerr = hipMemcpy(bDev, b, nb, hipMemcpyHostToDevice);

	for (int i = 0; i < n; i++)
	{
		a[i] = rand() * idrandmax;
		b[i] = rand() * idrandmax;
	}
	double startTime = getTime();
	for(int count=1; count<=20; count++)
	{
		int k=0;
		//power = power*10;

		//while(k<power)
		{
			int status = sum_host (argv[1], aDev, bDev, cDev, c, n);
			//if (status) return status;
			k++;
		}
		
	}
	double endTime = getTime();
	printf("%lf ", (endTime-startTime)/1000000.0);
	// Release device memory.
	if (aDev) hipFree(aDev);
	if (bDev) hipFree(bDev);
	if (cDev) hipFree(cDev);
	return 0;
}

