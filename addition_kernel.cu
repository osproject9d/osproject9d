
#include <hip/hip_runtime.h>

__global__ void addition_kernel(float* a, float* b, float* c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	c[idx] = a[idx] + b[idx];
}
/*
#define N (2048*2048)
#define THREADS_PER_BLOCK 512
__global__ void dot( int *a, int *b, int *c ) {
__shared__ int temp[THREADS_PER_BLOCK];
int index = threadIdx.x + blockIdx.x * blockDim.x;
temp[threadIdx.x] = a[index] * b[index];
__syncthreads();
if( 0 == threadIdx.x ) {
int sum = 0;
for( int i = 0; i < THREADS_PER_BLOCK; i++ )
sum += temp[i];
atomicAdd(
*c += sum; c , sum );
}
}
*/
