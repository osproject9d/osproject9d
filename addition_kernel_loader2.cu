#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int sum_host(const char* source, float* a, float* b, float* c, int n)
{
	int nb = n * sizeof ( float );
	float* aDev = NULL;
	float* bDev = NULL;
	float* cDev = NULL;
	
	int result = 0;	

	void* args[] = { (void*)&aDev, (void*)&bDev, (void*)&cDev };
	hipError_t cuerr;

	// Allocate memory on the GPU.
	hipError_t cudaerr = hipMalloc((void**)&aDev, nb);
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot allocate GPU memory for aDev: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}
	cudaerr = hipMalloc((void**)&bDev, nb);
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot allocate GPU memory for bDev: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}
	cudaerr = hipMalloc((void**)&cDev, nb);
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot allocate GPU memory for cDev: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}

	// Copy input data to device memory.
	cudaerr = hipMemcpy(aDev, a, nb, hipMemcpyHostToDevice);
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy data from a to aDev: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}
	cudaerr = hipMemcpy(bDev, b, nb, hipMemcpyHostToDevice);
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy data from b to bDev: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}

	// Load module.
	hipModule_t module;
	cuerr = hipModuleLoad(&module, source);
	assert(cuerr == hipSuccess);

	// Load kernel.
	hipFunction_t kernel;
	cuerr = hipModuleGetFunction(&kernel, module, "addition_kernel");
	assert(cuerr == hipSuccess);
	
	// Launch kernel.
	cuerr = hipModuleLaunchKernel(kernel,
		n / BLOCK_SIZE, 1, 1, BLOCK_SIZE, 1, 1, 512,
		0, args, NULL);
/*
		hipError_t hipModuleLaunchKernel	(hipFunction_t f,
		unsigned int  	gridDimX,
		unsigned int  	gridDimY,
		unsigned int  	gridDimZ,
		unsigned int  	blockDimX,
		unsigned int  	blockDimY,
		unsigned int  	blockDimZ,
		unsigned int  	sharedMemBytes,
		hipStream_t  	hStream,
		void **  	kernelParams,
		void **  	extra	 
	) 	
*/

	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot launch kernel: %d\n", cudaerr);
		result = 1;
		goto finish;
	}

	// Check error status from the launched kernel.
	cudaerr = hipGetLastError();
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}

	// Wait for kernel completion.
	cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}

	// Copy the resulting array back to the host memory.
	cudaerr = hipMemcpy(c, cDev, nb, hipMemcpyDeviceToHost);
	if (cudaerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy data from cdev to c: %s\n",
			hipGetErrorString(cudaerr));
		result = 1;
		goto finish;
	}

finish :

	// Release device memory.
	if (aDev) hipFree(aDev);
	if (bDev) hipFree(bDev);
	if (cDev) hipFree(cDev);

	return result;
}

#include <malloc.h>
#include <stdlib.h>

int main ( int argc, char* argv[] )
{
	if (argc != 3)
	{
		printf("Usage: %s <n> <source>\n", argv[0]);
		printf("Where n must be a multiplier of %d\n", BLOCK_SIZE);
		return 0;
	}

	int n = atoi(argv[1]), nb = n * sizeof(float);
	printf("n = %d\n", n);
	if (n <= 0)
	{
		fprintf(stderr, "Invalid n: %d, must be positive\n", n);
		return 1;
	}
	if (n % BLOCK_SIZE)
	{
		fprintf(stderr, "Invalid n: %d, must be a multiplier of %d\n",
			n, BLOCK_SIZE);
		return 1;
	}

	float* a = (float*)malloc(nb);
	float* b = (float*)malloc(nb);
	float* c = (float*)malloc(nb);
	double idrandmax = 1.0 / RAND_MAX;
	for (int i = 0; i < n; i++)
	{
		a[i] = rand() * idrandmax;
		b[i] = rand() * idrandmax;
	}

	int status = sum_host (argv[2], a, b, c, n);
	if (status) return status;

	int imaxdiff = 0;
	float maxdiff = 0.0;
	for (int i = 0; i < n; i++)
	{
		float diff = c[i] / (a[i] + b[i]);
		if (diff != diff) diff = 0; else diff = 1.0 - diff;
		if (diff > maxdiff)
		{
			maxdiff = diff;
			imaxdiff = i;
		}
	}
	
	printf("Max diff = %f @ i = %d: %f != %f\n",
		maxdiff * 100, imaxdiff, c[imaxdiff],
		a[imaxdiff] + b[imaxdiff]);
	return 0;
}

